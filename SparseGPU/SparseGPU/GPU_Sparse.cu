﻿#include <fstream>
#include <string>
#include <vector>
#include <iostream>
#include <sstream>
#include "hip/hip_runtime.h"


using namespace std;

int ReadNumbers(const string & s, vector <double> & v);
vector<double> matrixRead(const char* filename_X, vector <double>& v, int& rows, int& cols);
//cudaError gpuError(double* row_C, double* column_C, double* data, double* carpan, double* sonuc);
__global__ void gpu(double* row_C, double* column_C, double* data, double* x, double* y);
int main(){

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int rows = 0;
	int cols = 0;
	vector <double> vector_C;
	vector <double> vector_D;
	vector <double> vector_Rp;

	vector_C = matrixRead("C.txt", vector_C, rows, cols);
	//cout << "C vector :" << vector_C.size() << endl;
	vector_D = matrixRead("D.txt", vector_D, rows, cols);
	//cout << "D vector :" << vector_D.size() << endl;
	vector_Rp = matrixRead("Rp.txt", vector_Rp, rows, cols);
	//cout << "Rp vector :" << vector_Rp.size() << endl;

	vector <double> vector_carpan(10000);
	for (int i = 0; i <10000; i++)
		vector_carpan[i] = 1;
	//vector <double> gpuSonuc1(10000);

	double *vectorRow_p = &vector_Rp[0];
	double *vectorColumn_p = &vector_C[0];
	double *vectorData_p = &vector_D[0];
	double *vectorCarpan_p = &vector_carpan[0];
/*	double *gpuSonuc = &gpuSonuc1[0];

	cudaError_t cudaStatus = gpuError(vectorRow_p, vectorColumn_p, vectorData_p, vectorCarpan_p,gpuSonuc);
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		system("pause");
		return 1;
	}
	cudaStatus = cudaDeviceReset();
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaDeviceReset failed!");
		system("pause");
		return 1;
	}*/
	double *poutcome;
	poutcome = (double*)malloc(10000 * sizeof(double));
	memset(poutcome, 0, 10000 * sizeof(double));

	double *pRow, *pColumn, *pData, *pCarpan,*py;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	hipMalloc((void**)&pRow, 10001 * sizeof(double));
	hipMemcpy(pRow, vectorRow_p, 10001 * sizeof(double), hipMemcpyHostToDevice);

	hipMalloc((void**)&pColumn, 99911 * sizeof(double));
	hipMemcpy(pColumn, vectorColumn_p, 99911 * sizeof(double), hipMemcpyHostToDevice);

	hipMalloc((void**)&pData, 99911 * sizeof(double));
	hipMemcpy(pData, vectorData_p, 99911 * sizeof(double), hipMemcpyHostToDevice);

	hipMalloc((void**)&pCarpan, 10000 * sizeof(double));
	hipMemcpy(pCarpan,vectorCarpan_p, 10000 * sizeof(double), hipMemcpyHostToDevice);

	hipMalloc((void**)&py, 10000 * sizeof(double));

	
	hipEventRecord(start);
	gpu<<<1, 10000>>>(pRow, pColumn, pData, pCarpan, py);
	hipEventRecord(stop);
	hipMemcpy(poutcome, py, sizeof(double)* 10000, hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "Hesaplanan Sure(ms) : " << milliseconds << endl;


	free(poutcome);	
	hipFree(pRow);
	hipFree(pColumn);
	hipFree(pData);
	hipFree(pCarpan);
	hipFree(py);
	hipDeviceReset();
	
	system("pause");

	return 0;
}
/*cudaError gpuError(double* row_C, double* column_C, double* data, double* carpan, double* sonuc){
	double *vectorRow_p;
	double *vectorColumn_p ;
	double *vectorData_p ;
	double *vectorCarpan_p ;
	double *arraySonuc_p;
	double *gpuGelen_p = { 0 };
	cudaError_t cudaStatus;

	cudaStatus = cudaSetDevice(0);
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		exit(-1);
		system("pause");
	}
	cudaStatus = cudaMalloc((void**)&vectorRow_p, 10001 * sizeof(double));
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaMalloc failed!");
		exit(-1);
		system("pause");

	}
	cudaStatus = cudaMalloc((void**)&vectorColumn_p, 99911 * sizeof(double));
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaMalloc failed!");
		exit(-1);
		system("pause");

	}
	cudaStatus = cudaMalloc((void**)&vectorData_p, 99911 * sizeof(double));
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaMalloc failed!");
		exit(-1);
		system("pause");

	}
	cudaStatus = cudaMalloc((void**)&vectorCarpan_p, 10000 * sizeof(double));
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaMalloc failed!");
		exit(-1);
		system("pause");

	}
	cudaStatus = cudaMalloc((void**)&arraySonuc_p, 10000 * sizeof(double));
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaMalloc failed!");
		exit(-1);
		system("pause");

	}
	//-------------------------------------------------------------------------------------------//
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = cudaMemcpy(vectorRow_p, row_C, 10001 * sizeof(double), cudaMemcpyHostToDevice);
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaMemcpy failed!");
		exit(-1);
		system("pause");

	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = cudaMemcpy(vectorColumn_p, column_C, 99911 * sizeof(double), cudaMemcpyHostToDevice);
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaMemcpy failed!");
		exit(-1);
		system("pause");

	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = cudaMemcpy(vectorData_p, data, 99911 * sizeof(double), cudaMemcpyHostToDevice);
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaMemcpy failed!");
		exit(-1);
		system("pause");

	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = cudaMemcpy(vectorCarpan_p, carpan, 10000 * sizeof(double), cudaMemcpyHostToDevice);
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaMemcpy failed!");
		exit(-1);
		system("pause");

	}
	
	gpu << <1, 1000 >> >(vectorRow_p, vectorColumn_p, vectorData_p, vectorCarpan_p, arraySonuc_p);
	cudaStatus = cudaGetLastError();
	if (cudaStatus != cudaSuccess) {
		//fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
		cout << "addKernel launch failed:" << cudaGetErrorString(cudaStatus) << endl;
		system("pause");

	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = cudaDeviceSynchronize();
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		exit(-1);
		system("pause");

	}
	cudaStatus = cudaMemcpy(gpuGelen_p, arraySonuc_p, sizeof(double)* 1000, cudaMemcpyDeviceToHost);
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaMemcpy failed!");
		exit(-1);
		system("pause");

	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = cudaMemcpy(arraySonuc_p, sonuc, 10000 * sizeof(double), cudaMemcpyHostToDevice);
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaMemcpy failed!");
		exit(-1);
		system("pause");

	}
	
	cudaFree(vectorRow_p);
	cudaFree(vectorColumn_p);
	cudaFree(vectorData_p);
	cudaFree(vectorCarpan_p);
	cudaFree(arraySonuc_p);

	return cudaStatus;
}*/
__global__ void gpu(double* row_C, double* column_C, double* data, double* x, double* y)
{
	int row = threadIdx.x + blockIdx.x*blockDim.x;
	if (row< 10000)//son satira girmemek için
		{
			float carpan = 0;

			int row_start = row_C[row];
			int row_end = row_C[row + 1];
			for (int i = row_start; i < row_end; i++)
				carpan += data[i - 1];// *x[column_C[i]];
			y[row] = carpan;
			//printf("hello Word");
			//printf("%d\n", y[row]);
		}
	//cout << y[0] << endl;
}
int ReadNumbers(const string & s, vector <double> & v) {
	istringstream is(s);
	double n;
	while (is >> n) {
		v.push_back(n);
	}
	return v.size();
}
vector<double> matrixRead(const char* filename_X, vector <double>& vector, int& rows, int& cols){

	ifstream fileTxt;
	string line;

	fileTxt.open(filename_X);
	if (fileTxt.is_open())
	{
		int i = 0;
		getline(fileTxt, line);


		cols = ReadNumbers(line, vector);
		//cout << "cols:" << cols << endl;


		for (i = 1; i<99912; i++){
			if (getline(fileTxt, line).end) break;
			ReadNumbers(line, vector);

		}

		rows = i;
		//cout << "rows :" << rows << endl;
		if (rows >99912) cout << "N must be smaller than MAX_INT";

		fileTxt.close();
	}
	else{
		cout << "file open failed";
	}

	//cout << "vector:"<<endl;
	for (int i = 0; i<rows; i++){
		for (int j = 0; j<cols; j++){
			//	cout << vector[i*cols + j] << "\t";
		}
		//cout << vector[0]<<endl;
		//cout << endl;
	}
	return vector;
}
